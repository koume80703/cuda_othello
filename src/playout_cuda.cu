#include "header/playout_cuda.cuh"
#include "header/kernel.cuh"

double get_time_sec()
{
    return static_cast<double>(duration_cast<nanoseconds>(steady_clock::now().time_since_epoch()).count()) / (1000 * 1000 * 1000);
}

double get_time_msec()
{
    return static_cast<double>(duration_cast<nanoseconds>(steady_clock::now().time_since_epoch()).count()) / (1000 * 1000);
}

float playout_cuda(State state)
{
    double start, tmp, elapsed, total = 0;
    start = get_time_msec();

    int nElem = 4096;
    size_t size_sc = sizeof(STATE_CUDA);
    size_t size_result = nElem * sizeof(float);

    STATE_CUDA *h_sc, *d_sc;
    h_sc = (STATE_CUDA *)malloc(size_sc);
    trans_data(state, h_sc);

    CHECK(hipMalloc((STATE_CUDA **)&d_sc, size_sc));

    float *h_result, *d_result;
    h_result = (float *)malloc(size_result);
    memset(h_result, 0, nElem);
    CHECK(hipMalloc((float **)&d_result, size_result));

    CHECK(hipMemcpy(d_sc, h_sc, size_sc, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_result, h_result, size_result, hipMemcpyHostToDevice));

    const int threads_per_block = 1024;

    dim3 block(threads_per_block, 1, 1);
    dim3 grid(nElem / block.x, 1, 1);

    std::random_device rnd;
    int seed = rnd();

    tmp = get_time_msec();
    elapsed = tmp - start;
    start = tmp;
    printf("memory allocate time: %.3f [ms], ", elapsed);
    total += elapsed;

    kernel<<<grid, block>>>(d_sc, d_result, seed);

    CHECK(hipDeviceSynchronize());

    tmp = get_time_msec();
    elapsed = tmp - start;
    start = tmp;
    printf("kernel execute time: %.3f [ms], ", elapsed);
    total += elapsed;

    CHECK(hipMemcpy(h_result, d_result, size_result, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_sc));
    CHECK(hipFree(d_result));

    int sum_result = 0;
    for (int i = 0; i < nElem; i++)
    {
        sum_result += h_result[i];
    }

    free(h_sc);
    free(h_result);

    tmp = get_time_msec();
    elapsed = tmp - start;
    total += elapsed;

    printf("others time: %.3f [ms], total time: %.3f [ms]\n", elapsed, total);

    return sum_result;
}
